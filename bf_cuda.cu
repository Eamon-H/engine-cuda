#include "hip/hip_runtime.h"
#ifndef __DEVICE_EMULATION__

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <openssl/blowfish.h>
#include <hip/hip_runtime_api.h>
#include "cuda_common.h"
#include "common.h"
//#include "lib/cuPrintf.cu"

__constant__ BF_KEY bf_constant_schedule;
//__shared__ BF_KEY bf_schedule;

__device__ uint64_t *bf_device_data;
uint8_t  *bf_host_data;

float bf_elapsed;
hipEvent_t bf_start,bf_stop;

#define BF_M  (0xFF<<2)
#define BF_0  (24-2)
#define BF_1  (16-2)
#define BF_2  ( 8-2)
#define BF_3  2
#define BF_ENC(LL,R,S,P) ( \
	LL^=P, \
	LL^= (((*(BF_LONG *)((unsigned char *)&(S[  0])+((R>>BF_0)&BF_M))+ \
		*(BF_LONG *)((unsigned char *)&(S[256])+((R>>BF_1)&BF_M)))^ \
		*(BF_LONG *)((unsigned char *)&(S[512])+((R>>BF_2)&BF_M)))+ \
		*(BF_LONG *)((unsigned char *)&(S[768])+((R<<BF_3)&BF_M))) \
	)

__global__ void BFencKernel(uint64_t *data) {
	register uint32_t l, r;
	register uint64_t block = data[TX];

	// TODO: Let's see if this is really faster than constant memory!
	/*
	if(threadIdx.x < 18)
		bf_schedule.P[threadIdx.x] = bf_constant_schedule.P[threadIdx.x];

	bf_schedule.S[threadIdx.x] = bf_constant_schedule.S[threadIdx.x];
	bf_schedule.S[threadIdx.x+128] = bf_constant_schedule.S[threadIdx.x+128];
	bf_schedule.S[threadIdx.x+256] = bf_constant_schedule.S[threadIdx.x+256];
	bf_schedule.S[threadIdx.x+384] = bf_constant_schedule.S[threadIdx.x+384];
	bf_schedule.S[threadIdx.x+512] = bf_constant_schedule.S[threadIdx.x+512];
	bf_schedule.S[threadIdx.x+640] = bf_constant_schedule.S[threadIdx.x+640];
	bf_schedule.S[threadIdx.x+768] = bf_constant_schedule.S[threadIdx.x+768];
	bf_schedule.S[threadIdx.x+896] = bf_constant_schedule.S[threadIdx.x+896];
	// TODO: Let's see if synching isn't needed
	//__syncthreads();
	*/

	n2l((unsigned char *)&block,l);
	n2l(((unsigned char *)&block)+4,r);

	register const uint32_t *p,*s;

	p=&(bf_constant_schedule.P[0]);
	s=&(bf_constant_schedule.S[0]);

	l^=p[0];
	BF_ENC(r,l,s,p[ 1]);
	BF_ENC(l,r,s,p[ 2]);
	BF_ENC(r,l,s,p[ 3]);
	BF_ENC(l,r,s,p[ 4]);
	BF_ENC(r,l,s,p[ 5]);
	BF_ENC(l,r,s,p[ 6]);
	BF_ENC(r,l,s,p[ 7]);
	BF_ENC(l,r,s,p[ 8]);
	BF_ENC(r,l,s,p[ 9]);
	BF_ENC(l,r,s,p[10]);
	BF_ENC(r,l,s,p[11]);
	BF_ENC(l,r,s,p[12]);
	BF_ENC(r,l,s,p[13]);
	BF_ENC(l,r,s,p[14]);
	BF_ENC(r,l,s,p[15]);
	BF_ENC(l,r,s,p[16]);
	r^=p[BF_ROUNDS+1];

	block = ((uint64_t)r) << 32 | l;
	flip64(block);
	data[TX] = block;

}

__global__ void BFdecKernel(uint64_t *data) {
	
}

extern "C" void BF_cuda_crypt(const unsigned char *in, unsigned char *out, size_t nbytes, int enc, uint8_t **host_data, uint64_t **device_data) {
	assert(in && out && nbytes);
	hipError_t cudaerrno;
	int gridSize;
	dim3 dimBlock(MAX_THREAD, 1, 1);

	transferHostToDevice(&in, (uint32_t **)device_data, host_data, &nbytes);

	if ((nbytes%(MAX_THREAD*BF_BLOCK_SIZE))==0) {
		gridSize = nbytes/(MAX_THREAD*BF_BLOCK_SIZE);
	} else {
		if (nbytes < MAX_THREAD*BF_BLOCK_SIZE)
			dimBlock.x = nbytes / 8;
		gridSize = nbytes/(MAX_THREAD*BF_BLOCK_SIZE)+1;
	}

	if (output_verbosity==OUTPUT_VERBOSE)
		fprintf(stdout,"Starting BF kernel for %zu bytes with (%d, (%d, %d))...\n", nbytes, gridSize, dimBlock.x, dimBlock.y);

	if(enc == BF_ENCRYPT) {
		BFencKernel<<<gridSize,dimBlock>>>(*device_data);
		_CUDA_N("BF encryption kernel could not be launched!");
	} else {
		BFdecKernel<<<gridSize,dimBlock>>>(*device_data);
		_CUDA_N("BF decryption kernel could not be launched!");
	}

	transferDeviceToHost(&out, (uint32_t **)device_data, host_data, host_data, &nbytes);
}

extern "C" void BF_cuda_transfer_key_schedule(BF_KEY *ks) {
	assert(ks);
	hipError_t cudaerrno;
	size_t ks_size = sizeof(BF_KEY);
	_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(bf_constant_schedule),ks,ks_size,0,hipMemcpyHostToDevice));
}

//
// CBC parallel decrypt
//

__global__ void BFdecKernel_cbc(uint32_t in[],uint32_t out[],uint32_t iv[]) {
}

extern "C" void BF_cuda_transfer_iv(const unsigned char *iv) {
}

extern "C" void BF_cuda_decrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}

#ifndef CBC_ENC_CPU
//
// CBC  encrypt
//

__global__ void BFencKernel_cbc(uint32_t state[],uint32_t iv[],size_t length) {
}

#endif

extern "C" void BF_cuda_encrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}
#else
#error "ERROR: DEVICE EMULATION is NOT supported."
#endif
