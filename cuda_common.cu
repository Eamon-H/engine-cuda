#include <cuda_common.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <common.h>

#ifndef PAGEABLE
extern "C" void transferHostToDevice_PINNED (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	memcpy(*hostMem,*input,*size);
        _CUDA(hipMemcpyAsync(*deviceMem, *hostMem, *size, hipMemcpyHostToDevice, 0));
}
#if CUDART_VERSION >= 2020
extern "C" void transferHostToDevice_ZEROCOPY (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	memcpy(*hostMem,*input,*size);
}
#endif
#else
extern "C" void transferHostToDevice_PAGEABLE (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(*deviceMem, *input, *size, hipMemcpyHostToDevice));
}
#endif
#ifndef PAGEABLE
extern "C" void transferDeviceToHost_PINNED   (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
        _CUDA(hipMemcpyAsync(*hostMem, *deviceMem, *size, hipMemcpyDeviceToHost, 0));
	_CUDA(hipDeviceSynchronize());
	memcpy(*output,*hostMem,*size);
}
#if CUDART_VERSION >= 2020
extern "C" void transferDeviceToHost_ZEROCOPY (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipDeviceSynchronize());
	memcpy(*output,*hostMem,*size);
}
#endif
#else
extern "C" void transferDeviceToHost_PAGEABLE (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(*output,*deviceMem,*size, hipMemcpyDeviceToHost));
}
#endif

void checkCUDADevice(struct hipDeviceProp_t *deviceProp, int output_verbosity) {
	int deviceCount;
	hipError_t cudaerrno;

	_CUDA(hipGetDeviceCount(&deviceCount));

	if (!deviceCount) {
		if (output_verbosity!=OUTPUT_QUIET) 
			fprintf(stderr,"There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	} else {
		if (output_verbosity>=OUTPUT_NORMAL) 
			fprintf(stdout,"Successfully found %d CUDA devices (CUDART_VERSION %d).\n",deviceCount, CUDART_VERSION);
	}
	_CUDA(hipSetDevice(0));
	_CUDA(hipGetDeviceProperties(deviceProp, 0));
	
	if (output_verbosity==OUTPUT_VERBOSE) {
        	fprintf(stdout,"\nDevice %d: \"%s\"\n", 0, deviceProp->name);
      	 	fprintf(stdout,"  CUDA Compute Capability:                       %d.%d\n", deviceProp->major,deviceProp->minor);
#if CUDART_VERSION >= 2000
        	fprintf(stdout,"  Number of multiprocessors (SM):                %d\n", deviceProp->multiProcessorCount);
#endif
#if CUDART_VERSION >= 2020
		fprintf(stdout,"  Integrated:                                    %s\n", deviceProp->integrated ? "Yes" : "No");
        	fprintf(stdout,"  Support host page-locked memory mapping:       %s\n", deviceProp->canMapHostMemory ? "Yes" : "No");
#endif
		fprintf(stdout,"\n");
		}
}
