#include <cuda_common.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <common.h>
#include <assert.h>

#ifndef PAGEABLE
extern "C" void transferHostToDevice_PINNED   (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	memcpy(*hostMem,*input,*size);
        _CUDA(hipMemcpyAsync(*deviceMem, *hostMem, *size, hipMemcpyHostToDevice, 0));
}
#if CUDART_VERSION >= 2020
extern "C" void transferHostToDevice_ZEROCOPY (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	//hipError_t cudaerrno;
	memcpy(*hostMem,*input,*size);
	//_CUDA(hipHostGetDevicePointer(&d_s,h_s, 0));
}
#endif
#else
extern "C" void transferHostToDevice_PAGEABLE (const unsigned char **input, uint32_t **deviceMem, uint8_t **hostMem, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(*deviceMem, *input, *size, hipMemcpyHostToDevice));
}
#endif

#ifndef PAGEABLE
extern "C" void transferDeviceToHost_PINNED   (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMemS, uint8_t **hostMemOUT, size_t *size) {
	hipError_t cudaerrno;
        _CUDA(hipMemcpyAsync(*hostMemS, *deviceMem, *size, hipMemcpyDeviceToHost, 0));
	_CUDA(hipDeviceSynchronize());
	memcpy(*output,*hostMemS,*size);
}
#if CUDART_VERSION >= 2020
extern "C" void transferDeviceToHost_ZEROCOPY (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMemS, uint8_t **hostMemOUT, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipDeviceSynchronize());
	memcpy(*output,*hostMemOUT,*size);
}
#endif
#else
extern "C" void transferDeviceToHost_PAGEABLE (unsigned char **output, uint32_t **deviceMem, uint8_t **hostMemS, uint8_t **hostMemOUT, size_t *size) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpy(*output,*deviceMem,*size, hipMemcpyDeviceToHost));
}
#endif


float time_elapsed;
hipEvent_t time_start,time_stop;

void checkCUDADevice(struct hipDeviceProp_t *deviceProp, int output_verbosity) {
	int deviceCount;
	hipError_t cudaerrno;

	_CUDA(hipGetDeviceCount(&deviceCount));

	if (!deviceCount) {
		if (output_verbosity!=OUTPUT_QUIET) 
			fprintf(stderr,"There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	} else {
		if (output_verbosity>=OUTPUT_NORMAL) 
			fprintf(stdout,"Successfully found %d CUDA devices (CUDART_VERSION %d).\n",deviceCount, CUDART_VERSION);
	}
	_CUDA(hipSetDevice(0));
	_CUDA(hipGetDeviceProperties(deviceProp, 0));
	
	if (output_verbosity==OUTPUT_VERBOSE) {
        	fprintf(stdout,"\nDevice %d: \"%s\"\n", 0, deviceProp->name);
      	 	fprintf(stdout,"  CUDA Compute Capability:                       %d.%d\n", deviceProp->major,deviceProp->minor);
#if CUDART_VERSION >= 2000
        	fprintf(stdout,"  Number of multiprocessors (SM):                %d\n", deviceProp->multiProcessorCount);
#endif
#if CUDART_VERSION >= 2020
		fprintf(stdout,"  Integrated:                                    %s\n", deviceProp->integrated ? "Yes" : "No");
        	fprintf(stdout,"  Support host page-locked memory mapping:       %s\n", deviceProp->canMapHostMemory ? "Yes" : "No");
#endif
		fprintf(stdout,"\n");
		}
}

extern "C" void cuda_device_init(int *nm, int buffer_size, int output_verbosity, uint8_t **host_data, uint64_t **device_data) {
	assert(nm);
	hipError_t cudaerrno;
	hipDeviceProp_t deviceProp;
    	
	checkCUDADevice(&deviceProp, output_verbosity);
	
	if(buffer_size==0)
		buffer_size=MAX_CHUNK_SIZE;
	
#if CUDART_VERSION >= 2000
	*nm=deviceProp.multiProcessorCount;
#endif

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	isIntegrated=deviceProp.integrated;
	if(isIntegrated) {
        	//zero-copy memory mode - use special function to get OS-pinned memory
		_CUDA(hipSetDeviceFlags(hipDeviceMapHost));
        	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using zero-copy memory.\n");
        	_CUDA(hipHostAlloc((void**)host_data,buffer_size,hipHostMallocMapped));
		transferHostToDevice = transferHostToDevice_ZEROCOPY;		// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_ZEROCOPY;		// set memory transfer function
		_CUDA(hipHostGetDevicePointer(device_data,host_data, 0));
	} else {
		//pinned memory mode - use special function to get OS-pinned memory
		_CUDA(hipHostAlloc( (void**)host_data, buffer_size, hipHostMallocDefault));
		if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
		transferHostToDevice = transferHostToDevice_PINNED;	// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_PINNED;	// set memory transfer function
		_CUDA(hipMalloc((void **)device_data,buffer_size));
	}
#else
        //pinned memory mode - use special function to get OS-pinned memory
        _CUDA(hipHostMalloc((void**)&h_s, buffer_size));
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
	transferHostToDevice = transferHostToDevice_PINNED;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PINNED;			// set memory transfer function
	_CUDA(hipMalloc((void **)device_data,buffer_size));
#endif
#else
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pageable memory.\n");
	transferHostToDevice = transferHostToDevice_PAGEABLE;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PAGEABLE;			// set memory transfer function
	_CUDA(hipMalloc((void **)device_data,buffer_size));
#endif

	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"The current buffer size is %d.\n\n", buffer_size);

	_CUDA(hipEventCreate(&time_start));
	_CUDA(hipEventCreate(&time_stop));
	_CUDA(hipEventRecord(time_start,0));

}

extern "C" void cuda_device_finish(uint8_t *host_data, uint64_t *device_data) {
	hipError_t cudaerrno;

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout, "\nDone. Finishing up...\n");

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	if(isIntegrated) {
		_CUDA(hipHostFree(host_data));
		//_CUDA(hipHostFree(h_iv));
	} else {
		_CUDA(hipFree(device_data));
		//_CUDA(hipFree(d_iv));
	}
#else	
	_CUDA(hipFree(device_data));
	//_CUDA(hipFree(d_iv));
#endif
#else
	_CUDA(hipFree(device_data));
	//_CUDA(hipFree(d_iv));
#endif	

	_CUDA(hipEventRecord(time_stop,0));
	_CUDA(hipEventSynchronize(time_stop));
	_CUDA(hipEventElapsedTime(&time_elapsed,time_start,time_stop));

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout,"\nTotal time: %f milliseconds\n",time_elapsed);	
}
