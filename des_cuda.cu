#include "hip/hip_runtime.h"
#ifndef __DEVICE_EMULATION__

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <openssl/des.h>
#include "cuda_common.h"
#include "common.h"

__constant__ uint32_t des_d_sp_c[8][64]={
{
/* nibble 0 */
0x02080800L, 0x00080000L, 0x02000002L, 0x02080802L,
0x02000000L, 0x00080802L, 0x00080002L, 0x02000002L,
0x00080802L, 0x02080800L, 0x02080000L, 0x00000802L,
0x02000802L, 0x02000000L, 0x00000000L, 0x00080002L,
0x00080000L, 0x00000002L, 0x02000800L, 0x00080800L,
0x02080802L, 0x02080000L, 0x00000802L, 0x02000800L,
0x00000002L, 0x00000800L, 0x00080800L, 0x02080002L,
0x00000800L, 0x02000802L, 0x02080002L, 0x00000000L,
0x00000000L, 0x02080802L, 0x02000800L, 0x00080002L,
0x02080800L, 0x00080000L, 0x00000802L, 0x02000800L,
0x02080002L, 0x00000800L, 0x00080800L, 0x02000002L,
0x00080802L, 0x00000002L, 0x02000002L, 0x02080000L,
0x02080802L, 0x00080800L, 0x02080000L, 0x02000802L,
0x02000000L, 0x00000802L, 0x00080002L, 0x00000000L,
0x00080000L, 0x02000000L, 0x02000802L, 0x02080800L,
0x00000002L, 0x02080002L, 0x00000800L, 0x00080802L,
},{
/* nibble 1 */
0x40108010L, 0x00000000L, 0x00108000L, 0x40100000L,
0x40000010L, 0x00008010L, 0x40008000L, 0x00108000L,
0x00008000L, 0x40100010L, 0x00000010L, 0x40008000L,
0x00100010L, 0x40108000L, 0x40100000L, 0x00000010L,
0x00100000L, 0x40008010L, 0x40100010L, 0x00008000L,
0x00108010L, 0x40000000L, 0x00000000L, 0x00100010L,
0x40008010L, 0x00108010L, 0x40108000L, 0x40000010L,
0x40000000L, 0x00100000L, 0x00008010L, 0x40108010L,
0x00100010L, 0x40108000L, 0x40008000L, 0x00108010L,
0x40108010L, 0x00100010L, 0x40000010L, 0x00000000L,
0x40000000L, 0x00008010L, 0x00100000L, 0x40100010L,
0x00008000L, 0x40000000L, 0x00108010L, 0x40008010L,
0x40108000L, 0x00008000L, 0x00000000L, 0x40000010L,
0x00000010L, 0x40108010L, 0x00108000L, 0x40100000L,
0x40100010L, 0x00100000L, 0x00008010L, 0x40008000L,
0x40008010L, 0x00000010L, 0x40100000L, 0x00108000L,
},{
/* nibble 2 */
0x04000001L, 0x04040100L, 0x00000100L, 0x04000101L,
0x00040001L, 0x04000000L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00040000L, 0x04040000L, 0x00000001L,
0x04040101L, 0x00000101L, 0x00000001L, 0x04040001L,
0x00000000L, 0x00040001L, 0x04040100L, 0x00000100L,
0x00000101L, 0x04040101L, 0x00040000L, 0x04000001L,
0x04040001L, 0x04000100L, 0x00040101L, 0x04040000L,
0x00040100L, 0x00000000L, 0x04000000L, 0x00040101L,
0x04040100L, 0x00000100L, 0x00000001L, 0x00040000L,
0x00000101L, 0x00040001L, 0x04040000L, 0x04000101L,
0x00000000L, 0x04040100L, 0x00040100L, 0x04040001L,
0x00040001L, 0x04000000L, 0x04040101L, 0x00000001L,
0x00040101L, 0x04000001L, 0x04000000L, 0x04040101L,
0x00040000L, 0x04000100L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00000000L, 0x04040001L, 0x00000101L,
0x04000001L, 0x00040101L, 0x00000100L, 0x04040000L,
},{
/* nibble 3 */
0x00401008L, 0x10001000L, 0x00000008L, 0x10401008L,
0x00000000L, 0x10400000L, 0x10001008L, 0x00400008L,
0x10401000L, 0x10000008L, 0x10000000L, 0x00001008L,
0x10000008L, 0x00401008L, 0x00400000L, 0x10000000L,
0x10400008L, 0x00401000L, 0x00001000L, 0x00000008L,
0x00401000L, 0x10001008L, 0x10400000L, 0x00001000L,
0x00001008L, 0x00000000L, 0x00400008L, 0x10401000L,
0x10001000L, 0x10400008L, 0x10401008L, 0x00400000L,
0x10400008L, 0x00001008L, 0x00400000L, 0x10000008L,
0x00401000L, 0x10001000L, 0x00000008L, 0x10400000L,
0x10001008L, 0x00000000L, 0x00001000L, 0x00400008L,
0x00000000L, 0x10400008L, 0x10401000L, 0x00001000L,
0x10000000L, 0x10401008L, 0x00401008L, 0x00400000L,
0x10401008L, 0x00000008L, 0x10001000L, 0x00401008L,
0x00400008L, 0x00401000L, 0x10400000L, 0x10001008L,
0x00001008L, 0x10000000L, 0x10000008L, 0x10401000L,
},{
/* nibble 4 */
0x08000000L, 0x00010000L, 0x00000400L, 0x08010420L,
0x08010020L, 0x08000400L, 0x00010420L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x00010400L,
0x08000420L, 0x08010020L, 0x08010400L, 0x00000000L,
0x00010400L, 0x08000000L, 0x00010020L, 0x00000420L,
0x08000400L, 0x00010420L, 0x00000000L, 0x08000020L,
0x00000020L, 0x08000420L, 0x08010420L, 0x00010020L,
0x08010000L, 0x00000400L, 0x00000420L, 0x08010400L,
0x08010400L, 0x08000420L, 0x00010020L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x08000400L,
0x08000000L, 0x00010400L, 0x08010420L, 0x00000000L,
0x00010420L, 0x08000000L, 0x00000400L, 0x00010020L,
0x08000420L, 0x00000400L, 0x00000000L, 0x08010420L,
0x08010020L, 0x08010400L, 0x00000420L, 0x00010000L,
0x00010400L, 0x08010020L, 0x08000400L, 0x00000420L,
0x00000020L, 0x00010420L, 0x08010000L, 0x08000020L,
},{
/* nibble 5 */
0x80000040L, 0x00200040L, 0x00000000L, 0x80202000L,
0x00200040L, 0x00002000L, 0x80002040L, 0x00200000L,
0x00002040L, 0x80202040L, 0x00202000L, 0x80000000L,
0x80002000L, 0x80000040L, 0x80200000L, 0x00202040L,
0x00200000L, 0x80002040L, 0x80200040L, 0x00000000L,
0x00002000L, 0x00000040L, 0x80202000L, 0x80200040L,
0x80202040L, 0x80200000L, 0x80000000L, 0x00002040L,
0x00000040L, 0x00202000L, 0x00202040L, 0x80002000L,
0x00002040L, 0x80000000L, 0x80002000L, 0x00202040L,
0x80202000L, 0x00200040L, 0x00000000L, 0x80002000L,
0x80000000L, 0x00002000L, 0x80200040L, 0x00200000L,
0x00200040L, 0x80202040L, 0x00202000L, 0x00000040L,
0x80202040L, 0x00202000L, 0x00200000L, 0x80002040L,
0x80000040L, 0x80200000L, 0x00202040L, 0x00000000L,
0x00002000L, 0x80000040L, 0x80002040L, 0x80202000L,
0x80200000L, 0x00002040L, 0x00000040L, 0x80200040L,
},{
/* nibble 6 */
0x00004000L, 0x00000200L, 0x01000200L, 0x01000004L,
0x01004204L, 0x00004004L, 0x00004200L, 0x00000000L,
0x01000000L, 0x01000204L, 0x00000204L, 0x01004000L,
0x00000004L, 0x01004200L, 0x01004000L, 0x00000204L,
0x01000204L, 0x00004000L, 0x00004004L, 0x01004204L,
0x00000000L, 0x01000200L, 0x01000004L, 0x00004200L,
0x01004004L, 0x00004204L, 0x01004200L, 0x00000004L,
0x00004204L, 0x01004004L, 0x00000200L, 0x01000000L,
0x00004204L, 0x01004000L, 0x01004004L, 0x00000204L,
0x00004000L, 0x00000200L, 0x01000000L, 0x01004004L,
0x01000204L, 0x00004204L, 0x00004200L, 0x00000000L,
0x00000200L, 0x01000004L, 0x00000004L, 0x01000200L,
0x00000000L, 0x01000204L, 0x01000200L, 0x00004200L,
0x00000204L, 0x00004000L, 0x01004204L, 0x01000000L,
0x01004200L, 0x00000004L, 0x00004004L, 0x01004204L,
0x01000004L, 0x01004200L, 0x01004000L, 0x00004004L,
},{
/* nibble 7 */
0x20800080L, 0x20820000L, 0x00020080L, 0x00000000L,
0x20020000L, 0x00800080L, 0x20800000L, 0x20820080L,
0x00000080L, 0x20000000L, 0x00820000L, 0x00020080L,
0x00820080L, 0x20020080L, 0x20000080L, 0x20800000L,
0x00020000L, 0x00820080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x20000080L, 0x00000000L, 0x00820000L,
0x20000000L, 0x00800000L, 0x20020080L, 0x20800080L,
0x00800000L, 0x00020000L, 0x20820000L, 0x00000080L,
0x00800000L, 0x00020000L, 0x20000080L, 0x20820080L,
0x00020080L, 0x20000000L, 0x00000000L, 0x00820000L,
0x20800080L, 0x20020080L, 0x20020000L, 0x00800080L,
0x20820000L, 0x00000080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x00800000L, 0x20800000L, 0x20000080L,
0x00820000L, 0x00020080L, 0x20020080L, 0x20800000L,
0x00000080L, 0x20820000L, 0x00820080L, 0x00000000L,
0x20000000L, 0x20800080L, 0x00020000L, 0x00820080L,
}};

__shared__ uint32_t des_d_sp[8][64];

__constant__ uint64_t cs[16];
__shared__ uint64_t s[16];

//__device__ uint32_t *des_d_iv;

float des_elapsed;
hipEvent_t des_start,des_stop;

#define IP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(right,left,tt, 4,0x0f0f0f0fL); \
	PERM_OP(left,right,tt,16,0x0000ffffL); \
	PERM_OP(right,left,tt, 2,0x33333333L); \
	PERM_OP(left,right,tt, 8,0x00ff00ffL); \
	PERM_OP(right,left,tt, 1,0x55555555L); \
	}

#define FP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(left,right,tt, 1,0x55555555L); \
	PERM_OP(right,left,tt, 8,0x00ff00ffL); \
	PERM_OP(left,right,tt, 2,0x33333333L); \
	PERM_OP(right,left,tt,16,0x0000ffffL); \
	PERM_OP(left,right,tt, 4,0x0f0f0f0fL); \
	}

#define	ROTATE(a,n)	(((a)>>(n))|((a)<<(32-(n))))

#define PERM_OP(a,b,t,n,m) ((t)=((((a)>>(n))^(b))&(m)),\
	(b)^=(t),\
	(a)^=((t)<<(n)))

#define D_ENCRYPT(LL,R,S) { \
	register uint64_t ss = s[S]; \
	u=R^ss; \
	t=R^ss>>32; \
	t=ROTATE(t,4); \
	LL^= \
	*(const uint32_t *)(des_SP      +((u     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x200+((u>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x400+((u>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x600+((u>>24L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x100+((t     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x300+((t>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x500+((t>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x700+((t>>24L)&0xfc)); }

__global__ void DESencKernel(uint64_t *data) {
	
	//if(threadIdx.x < 16)
		s[threadIdx.x%16] = cs[threadIdx.x%16];
	//__syncthreads();

	// Careful: Based on the assumption of a constant 128 threads!
	// What happens for kernel calls with less than 128 threads, like the final padding 8-byte call?
	// It seems to work, but might be because of a strange race condition. Watch out!
	((uint64_t *)des_d_sp)[threadIdx.x] = ((uint64_t *)des_d_sp_c)[threadIdx.x];
	((uint64_t *)des_d_sp)[threadIdx.x+128] = ((uint64_t *)des_d_sp_c)[threadIdx.x+128];

	register uint64_t load = data[TX];
	register uint32_t right = load;
	register uint32_t left = load >> 32;
	
	unsigned int t,u;
	unsigned char *des_SP = (unsigned char *) (&des_d_sp);

	IP(right,left);

	left=ROTATE(left,29);
	right=ROTATE(right,29);

	D_ENCRYPT(left,right, 0);
	D_ENCRYPT(right,left, 1);
	D_ENCRYPT(left,right, 2);
	D_ENCRYPT(right,left, 3);
	D_ENCRYPT(left,right, 4);
	D_ENCRYPT(right,left, 5);
	D_ENCRYPT(left,right, 6);
	D_ENCRYPT(right,left, 7);
	D_ENCRYPT(left,right, 8);
	D_ENCRYPT(right,left, 9);
	D_ENCRYPT(left,right,10);
	D_ENCRYPT(right,left,11);
	D_ENCRYPT(left,right,12);
	D_ENCRYPT(right,left,13);
	D_ENCRYPT(left,right,14);
	D_ENCRYPT(right,left,15);

	left=ROTATE(left,3);
	right=ROTATE(right,3);

	FP(right,left);
	load = left|((uint64_t)right)<<32;
	data[TX]=load;
}

__global__ void DESdecKernel(uint64_t *data) {
	
	if(threadIdx.x < 16)
		s[threadIdx.x] = cs[threadIdx.x];

	((uint64_t *)des_d_sp)[threadIdx.x] = ((uint64_t *)des_d_sp_c)[threadIdx.x];
	((uint64_t *)des_d_sp)[threadIdx.x+128] = ((uint64_t *)des_d_sp_c)[threadIdx.x+128];

	//uint64_t load = data[TX];
	uint32_t right = data[TX];
	uint32_t left = data[TX]>>32;

	unsigned int t,u;
	unsigned char *des_SP = (unsigned char *) (&des_d_sp);

	IP(right,left);

	left=ROTATE(left,29);
	right=ROTATE(right,29);

	D_ENCRYPT(left,right,15); /*  16 */
	D_ENCRYPT(right,left,14); /*  15 */
	D_ENCRYPT(left,right,13); /*  14 */
	D_ENCRYPT(right,left,12); /*  13 */
	D_ENCRYPT(left,right,11); /*  12 */
	D_ENCRYPT(right,left,10); /*  11 */
	D_ENCRYPT(left,right, 9); /*  10 */
	D_ENCRYPT(right,left, 8); /*  9 */
	D_ENCRYPT(left,right, 7); /*  8 */
	D_ENCRYPT(right,left, 6); /*  7 */
	D_ENCRYPT(left,right, 5); /*  6 */
	D_ENCRYPT(right,left, 4); /*  5 */
	D_ENCRYPT(left,right, 3); /*  4 */
	D_ENCRYPT(right,left, 2); /*  3 */
	D_ENCRYPT(left,right, 1); /*  2 */
	D_ENCRYPT(right,left, 0); /*  1 */

	left=ROTATE(left,3);
	right=ROTATE(right,3);

	FP(right,left);
	data[TX]=left|((uint64_t)right)<<32;
}

extern "C" void DES_cuda_crypt(const unsigned char *in, unsigned char *out, size_t nbytes, int enc, uint8_t **host_data, uint64_t **device_data) {
	assert(in && out && nbytes);
	hipError_t cudaerrno;
	int gridSize;

	transferHostToDevice(&in, (uint32_t **)device_data, host_data, &nbytes);
	
	if ((nbytes%(MAX_THREAD*DES_BLOCK_SIZE))==0) {
		gridSize = nbytes/(MAX_THREAD*DES_BLOCK_SIZE);
	} else {
		gridSize = nbytes/(MAX_THREAD*DES_BLOCK_SIZE)+1;
	}

	#ifdef DEBUG
		fprintf(stdout,"Starting DES kernel for %zu bytes with (%d, (%d))...\n", nbytes, gridSize, MAX_THREAD);
	#endif

	if(enc == DES_ENCRYPT) {
		DESencKernel<<<gridSize,MAX_THREAD>>>(*device_data);
		_CUDA_N("DES encryption kernel could not be launched!");
	} else {
		DESdecKernel<<<gridSize,MAX_THREAD>>>(*device_data);
		_CUDA_N("DES decryption kernel could not be launched!");
	}

	transferDeviceToHost(&out, (uint32_t **)device_data, host_data, host_data, &nbytes);
}

extern "C" void DES_cuda_transfer_key_schedule(DES_key_schedule *ks) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(cs),ks,sizeof(DES_key_schedule),0,hipMemcpyHostToDevice));
}

//
// CBC parallel decrypt
//

__global__ void DESdecKernel_cbc(uint32_t in[],uint32_t out[],uint32_t iv[]) {
}

extern "C" void DES_cuda_transfer_iv(const unsigned char *iv) {
}

extern "C" void DES_cuda_decrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}

#ifndef CBC_ENC_CPU
//
// CBC  encrypt
//

__global__ void DESencKernel_cbc(uint32_t state[],uint32_t iv[],size_t length) {
}

#endif

extern "C" void DES_cuda_encrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}
#else
#error "ERROR: DEVICE EMULATION is NOT supported."
#endif
