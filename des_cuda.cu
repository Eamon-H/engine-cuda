#include "hip/hip_runtime.h"
#ifndef __DEVICE_EMULATION__

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <openssl/des.h>
#include "cuda_common.h"
#include "common.h"
#include "lib/cuPrintf.cu"

__constant__ uint32_t des_d_sp_c[8][64]={
{
/* nibble 0 */
0x02080800L, 0x00080000L, 0x02000002L, 0x02080802L,
0x02000000L, 0x00080802L, 0x00080002L, 0x02000002L,
0x00080802L, 0x02080800L, 0x02080000L, 0x00000802L,
0x02000802L, 0x02000000L, 0x00000000L, 0x00080002L,
0x00080000L, 0x00000002L, 0x02000800L, 0x00080800L,
0x02080802L, 0x02080000L, 0x00000802L, 0x02000800L,
0x00000002L, 0x00000800L, 0x00080800L, 0x02080002L,
0x00000800L, 0x02000802L, 0x02080002L, 0x00000000L,
0x00000000L, 0x02080802L, 0x02000800L, 0x00080002L,
0x02080800L, 0x00080000L, 0x00000802L, 0x02000800L,
0x02080002L, 0x00000800L, 0x00080800L, 0x02000002L,
0x00080802L, 0x00000002L, 0x02000002L, 0x02080000L,
0x02080802L, 0x00080800L, 0x02080000L, 0x02000802L,
0x02000000L, 0x00000802L, 0x00080002L, 0x00000000L,
0x00080000L, 0x02000000L, 0x02000802L, 0x02080800L,
0x00000002L, 0x02080002L, 0x00000800L, 0x00080802L,
},{
/* nibble 1 */
0x40108010L, 0x00000000L, 0x00108000L, 0x40100000L,
0x40000010L, 0x00008010L, 0x40008000L, 0x00108000L,
0x00008000L, 0x40100010L, 0x00000010L, 0x40008000L,
0x00100010L, 0x40108000L, 0x40100000L, 0x00000010L,
0x00100000L, 0x40008010L, 0x40100010L, 0x00008000L,
0x00108010L, 0x40000000L, 0x00000000L, 0x00100010L,
0x40008010L, 0x00108010L, 0x40108000L, 0x40000010L,
0x40000000L, 0x00100000L, 0x00008010L, 0x40108010L,
0x00100010L, 0x40108000L, 0x40008000L, 0x00108010L,
0x40108010L, 0x00100010L, 0x40000010L, 0x00000000L,
0x40000000L, 0x00008010L, 0x00100000L, 0x40100010L,
0x00008000L, 0x40000000L, 0x00108010L, 0x40008010L,
0x40108000L, 0x00008000L, 0x00000000L, 0x40000010L,
0x00000010L, 0x40108010L, 0x00108000L, 0x40100000L,
0x40100010L, 0x00100000L, 0x00008010L, 0x40008000L,
0x40008010L, 0x00000010L, 0x40100000L, 0x00108000L,
},{
/* nibble 2 */
0x04000001L, 0x04040100L, 0x00000100L, 0x04000101L,
0x00040001L, 0x04000000L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00040000L, 0x04040000L, 0x00000001L,
0x04040101L, 0x00000101L, 0x00000001L, 0x04040001L,
0x00000000L, 0x00040001L, 0x04040100L, 0x00000100L,
0x00000101L, 0x04040101L, 0x00040000L, 0x04000001L,
0x04040001L, 0x04000100L, 0x00040101L, 0x04040000L,
0x00040100L, 0x00000000L, 0x04000000L, 0x00040101L,
0x04040100L, 0x00000100L, 0x00000001L, 0x00040000L,
0x00000101L, 0x00040001L, 0x04040000L, 0x04000101L,
0x00000000L, 0x04040100L, 0x00040100L, 0x04040001L,
0x00040001L, 0x04000000L, 0x04040101L, 0x00000001L,
0x00040101L, 0x04000001L, 0x04000000L, 0x04040101L,
0x00040000L, 0x04000100L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00000000L, 0x04040001L, 0x00000101L,
0x04000001L, 0x00040101L, 0x00000100L, 0x04040000L,
},{
/* nibble 3 */
0x00401008L, 0x10001000L, 0x00000008L, 0x10401008L,
0x00000000L, 0x10400000L, 0x10001008L, 0x00400008L,
0x10401000L, 0x10000008L, 0x10000000L, 0x00001008L,
0x10000008L, 0x00401008L, 0x00400000L, 0x10000000L,
0x10400008L, 0x00401000L, 0x00001000L, 0x00000008L,
0x00401000L, 0x10001008L, 0x10400000L, 0x00001000L,
0x00001008L, 0x00000000L, 0x00400008L, 0x10401000L,
0x10001000L, 0x10400008L, 0x10401008L, 0x00400000L,
0x10400008L, 0x00001008L, 0x00400000L, 0x10000008L,
0x00401000L, 0x10001000L, 0x00000008L, 0x10400000L,
0x10001008L, 0x00000000L, 0x00001000L, 0x00400008L,
0x00000000L, 0x10400008L, 0x10401000L, 0x00001000L,
0x10000000L, 0x10401008L, 0x00401008L, 0x00400000L,
0x10401008L, 0x00000008L, 0x10001000L, 0x00401008L,
0x00400008L, 0x00401000L, 0x10400000L, 0x10001008L,
0x00001008L, 0x10000000L, 0x10000008L, 0x10401000L,
},{
/* nibble 4 */
0x08000000L, 0x00010000L, 0x00000400L, 0x08010420L,
0x08010020L, 0x08000400L, 0x00010420L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x00010400L,
0x08000420L, 0x08010020L, 0x08010400L, 0x00000000L,
0x00010400L, 0x08000000L, 0x00010020L, 0x00000420L,
0x08000400L, 0x00010420L, 0x00000000L, 0x08000020L,
0x00000020L, 0x08000420L, 0x08010420L, 0x00010020L,
0x08010000L, 0x00000400L, 0x00000420L, 0x08010400L,
0x08010400L, 0x08000420L, 0x00010020L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x08000400L,
0x08000000L, 0x00010400L, 0x08010420L, 0x00000000L,
0x00010420L, 0x08000000L, 0x00000400L, 0x00010020L,
0x08000420L, 0x00000400L, 0x00000000L, 0x08010420L,
0x08010020L, 0x08010400L, 0x00000420L, 0x00010000L,
0x00010400L, 0x08010020L, 0x08000400L, 0x00000420L,
0x00000020L, 0x00010420L, 0x08010000L, 0x08000020L,
},{
/* nibble 5 */
0x80000040L, 0x00200040L, 0x00000000L, 0x80202000L,
0x00200040L, 0x00002000L, 0x80002040L, 0x00200000L,
0x00002040L, 0x80202040L, 0x00202000L, 0x80000000L,
0x80002000L, 0x80000040L, 0x80200000L, 0x00202040L,
0x00200000L, 0x80002040L, 0x80200040L, 0x00000000L,
0x00002000L, 0x00000040L, 0x80202000L, 0x80200040L,
0x80202040L, 0x80200000L, 0x80000000L, 0x00002040L,
0x00000040L, 0x00202000L, 0x00202040L, 0x80002000L,
0x00002040L, 0x80000000L, 0x80002000L, 0x00202040L,
0x80202000L, 0x00200040L, 0x00000000L, 0x80002000L,
0x80000000L, 0x00002000L, 0x80200040L, 0x00200000L,
0x00200040L, 0x80202040L, 0x00202000L, 0x00000040L,
0x80202040L, 0x00202000L, 0x00200000L, 0x80002040L,
0x80000040L, 0x80200000L, 0x00202040L, 0x00000000L,
0x00002000L, 0x80000040L, 0x80002040L, 0x80202000L,
0x80200000L, 0x00002040L, 0x00000040L, 0x80200040L,
},{
/* nibble 6 */
0x00004000L, 0x00000200L, 0x01000200L, 0x01000004L,
0x01004204L, 0x00004004L, 0x00004200L, 0x00000000L,
0x01000000L, 0x01000204L, 0x00000204L, 0x01004000L,
0x00000004L, 0x01004200L, 0x01004000L, 0x00000204L,
0x01000204L, 0x00004000L, 0x00004004L, 0x01004204L,
0x00000000L, 0x01000200L, 0x01000004L, 0x00004200L,
0x01004004L, 0x00004204L, 0x01004200L, 0x00000004L,
0x00004204L, 0x01004004L, 0x00000200L, 0x01000000L,
0x00004204L, 0x01004000L, 0x01004004L, 0x00000204L,
0x00004000L, 0x00000200L, 0x01000000L, 0x01004004L,
0x01000204L, 0x00004204L, 0x00004200L, 0x00000000L,
0x00000200L, 0x01000004L, 0x00000004L, 0x01000200L,
0x00000000L, 0x01000204L, 0x01000200L, 0x00004200L,
0x00000204L, 0x00004000L, 0x01004204L, 0x01000000L,
0x01004200L, 0x00000004L, 0x00004004L, 0x01004204L,
0x01000004L, 0x01004200L, 0x01004000L, 0x00004004L,
},{
/* nibble 7 */
0x20800080L, 0x20820000L, 0x00020080L, 0x00000000L,
0x20020000L, 0x00800080L, 0x20800000L, 0x20820080L,
0x00000080L, 0x20000000L, 0x00820000L, 0x00020080L,
0x00820080L, 0x20020080L, 0x20000080L, 0x20800000L,
0x00020000L, 0x00820080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x20000080L, 0x00000000L, 0x00820000L,
0x20000000L, 0x00800000L, 0x20020080L, 0x20800080L,
0x00800000L, 0x00020000L, 0x20820000L, 0x00000080L,
0x00800000L, 0x00020000L, 0x20000080L, 0x20820080L,
0x00020080L, 0x20000000L, 0x00000000L, 0x00820000L,
0x20800080L, 0x20020080L, 0x20020000L, 0x00800080L,
0x20820000L, 0x00000080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x00800000L, 0x20800000L, 0x20000080L,
0x00820000L, 0x00020080L, 0x20020080L, 0x20800000L,
0x00000080L, 0x20820000L, 0x00820080L, 0x00000000L,
0x20000000L, 0x20800080L, 0x00020000L, 0x00820080L,
}};

__shared__ uint32_t des_d_sp[8][64];

__constant__ uint64_t cs[16];
__shared__ uint64_t s[16];

__device__ __constant__ uint64_t *des_d_s;
uint8_t  *des_h_s;
//__device__ uint32_t *des_d_iv;

float des_elapsed;
hipEvent_t des_start,des_stop;

#define IP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(right,left,tt, 4,0x0f0f0f0fL); \
	PERM_OP(left,right,tt,16,0x0000ffffL); \
	PERM_OP(right,left,tt, 2,0x33333333L); \
	PERM_OP(left,right,tt, 8,0x00ff00ffL); \
	PERM_OP(right,left,tt, 1,0x55555555L); \
	}

#define FP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(left,right,tt, 1,0x55555555L); \
	PERM_OP(right,left,tt, 8,0x00ff00ffL); \
	PERM_OP(left,right,tt, 2,0x33333333L); \
	PERM_OP(right,left,tt,16,0x0000ffffL); \
	PERM_OP(left,right,tt, 4,0x0f0f0f0fL); \
	}

#define	ROTATE(a,n)	(((a)>>(n))|((a)<<(32-(n))))

#define PERM_OP(a,b,t,n,m) ((t)=((((a)>>(n))^(b))&(m)),\
	(b)^=(t),\
	(a)^=((t)<<(n)))

#define D_ENCRYPT(LL,R,S) { \
	register uint64_t ss = s[S]; \
	u=R^ss; \
	t=R^ss>>32; \
	t=ROTATE(t,4); \
	LL^= \
	*(const uint32_t *)(des_SP      +((u     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x200+((u>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x400+((u>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x600+((u>>24L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x100+((t     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x300+((t>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x500+((t>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x700+((t>>24L)&0xfc)); }

__global__ void DESencKernel(uint64_t *data) {
	
	if(threadIdx.x < 16)
		s[threadIdx.x] = cs[threadIdx.x];

	// Careful: Based on the assumption of a constant 128 threads!
	((uint32_t *)des_d_sp)[threadIdx.x] = ((uint32_t *)des_d_sp_c)[threadIdx.x];
	((uint32_t *)des_d_sp)[threadIdx.x+128] = ((uint32_t *)des_d_sp_c)[threadIdx.x+128];
	((uint32_t *)des_d_sp)[threadIdx.x+256] = ((uint32_t *)des_d_sp_c)[threadIdx.x+256];
	((uint32_t *)des_d_sp)[threadIdx.x+384] = ((uint32_t *)des_d_sp_c)[threadIdx.x+384];

	uint64_t load = data[TX];
	uint32_t right = load;
	uint32_t left = load>>32;
	
	unsigned int t,u;
	unsigned char *des_SP = (unsigned char *) (&des_d_sp);

	IP(right,left);

	left=ROTATE(left,29);
	right=ROTATE(right,29);

	D_ENCRYPT(left,right, 0);
	D_ENCRYPT(right,left, 1);
	D_ENCRYPT(left,right, 2);
	D_ENCRYPT(right,left, 3);
	D_ENCRYPT(left,right, 4);
	D_ENCRYPT(right,left, 5);
	D_ENCRYPT(left,right, 6);
	D_ENCRYPT(right,left, 7);
	D_ENCRYPT(left,right, 8);
	D_ENCRYPT(right,left, 9);
	D_ENCRYPT(left,right,10);
	D_ENCRYPT(right,left,11);
	D_ENCRYPT(left,right,12);
	D_ENCRYPT(right,left,13);
	D_ENCRYPT(left,right,14);
	D_ENCRYPT(right,left,15);

	left=ROTATE(left,3);
	right=ROTATE(right,3);

	FP(right,left);
	data[TX]=left|((uint64_t)right)<<32;
}

__global__ void DESdecKernel(uint64_t *data) {
	
	if(threadIdx.x < 16)
		s[threadIdx.x] = cs[threadIdx.x];

	((uint32_t *)des_d_sp)[threadIdx.x] = ((uint32_t *)des_d_sp_c)[threadIdx.x];
	((uint32_t *)des_d_sp)[threadIdx.x+128] = ((uint32_t *)des_d_sp_c)[threadIdx.x+128];
	((uint32_t *)des_d_sp)[threadIdx.x+256] = ((uint32_t *)des_d_sp_c)[threadIdx.x+256];
	((uint32_t *)des_d_sp)[threadIdx.x+384] = ((uint32_t *)des_d_sp_c)[threadIdx.x+384];

	uint64_t load = data[TX];
	uint32_t right = load;
	uint32_t left = load>>32;

	unsigned int t,u;
	unsigned char *des_SP = (unsigned char *) (&des_d_sp);

	IP(right,left);

	left=ROTATE(left,29);
	right=ROTATE(right,29);

	D_ENCRYPT(left,right,15); /*  16 */
	D_ENCRYPT(right,left,14); /*  15 */
	D_ENCRYPT(left,right,13); /*  14 */
	D_ENCRYPT(right,left,12); /*  13 */
	D_ENCRYPT(left,right,11); /*  12 */
	D_ENCRYPT(right,left,10); /*  11 */
	D_ENCRYPT(left,right, 9); /*  10 */
	D_ENCRYPT(right,left, 8); /*  9 */
	D_ENCRYPT(left,right, 7); /*  8 */
	D_ENCRYPT(right,left, 6); /*  7 */
	D_ENCRYPT(left,right, 5); /*  6 */
	D_ENCRYPT(right,left, 4); /*  5 */
	D_ENCRYPT(left,right, 3); /*  4 */
	D_ENCRYPT(right,left, 2); /*  3 */
	D_ENCRYPT(left,right, 1); /*  2 */
	D_ENCRYPT(right,left, 0); /*  1 */

	left=ROTATE(left,3);
	right=ROTATE(right,3);

	FP(right,left);
	data[TX]=left|((uint64_t)right)<<32;
}

extern "C" void DES_cuda_crypt(const unsigned char *in, unsigned char *out, size_t nbytes, int enc) {
	assert(in && out && nbytes);
	hipError_t cudaerrno;
	int gridSize;
	dim3 dimBlock(MAX_THREAD, 1, 1);

	transferHostToDevice(&in, (uint32_t **)&des_d_s, &des_h_s, &nbytes);

	if ((nbytes%(MAX_THREAD*DES_BLOCK_SIZE))==0) {
		gridSize = nbytes/(MAX_THREAD*2);
	} else {
		gridSize = nbytes/(MAX_THREAD*2)+1;
	}

	if (output_verbosity==OUTPUT_VERBOSE)
		fprintf(stdout,"Starting DES kernel with (%d, (%d, %d))...\n", gridSize, dimBlock.x, dimBlock.y);

	if(enc == DES_ENCRYPT) {
		DESencKernel<<<gridSize,dimBlock>>>(des_d_s);
		_CUDA_N("DES encryption kernel could not be launched!");
	} else {
		DESdecKernel<<<gridSize,dimBlock>>>(des_d_s);
		_CUDA_N("DES decryption kernel could not be launched!");
	}

	transferDeviceToHost(&out, (uint32_t **)&des_d_s, &des_h_s, &nbytes);
}

extern "C" void DES_cuda_transfer_key_schedule(DES_key_schedule *ks) {
	assert(ks);
	hipError_t cudaerrno;
	size_t ks_size = sizeof(DES_key_schedule);
	_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(cs),ks,ks_size,0,hipMemcpyHostToDevice));
}

extern "C" void DES_cuda_finish() {
	hipError_t cudaerrno;

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout, "\nDone. Finishing up DES\n");

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	if(isIntegrated) {
		_CUDA(hipHostFree(des_h_s));
		//_CUDA(hipHostFree(des_h_iv));
	} else {
		_CUDA(hipFree(des_d_s));
		//_CUDA(hipFree(des_d_iv));
	}
#else	
	_CUDA(hipFree(des_d_s));
	//_CUDA(hipFree(des_d_iv));
#endif
#else
	_CUDA(hipFree(des_d_s));
	//_CUDA(hipFree(des_d_iv));
#endif	

	_CUDA(hipEventRecord(des_stop,0));
	_CUDA(hipEventSynchronize(des_stop));
	_CUDA(hipEventElapsedTime(&des_elapsed,des_start,des_stop));

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout,"\nTotal time: %f milliseconds\n",des_elapsed);	
}

extern "C" void DES_cuda_init(int *nm, int buffer_size_engine, int output_kind) {
	assert(nm);
	hipError_t cudaerrno;
   	int buffer_size;
	hipDeviceProp_t deviceProp;
    	
	output_verbosity=output_kind;

	checkCUDADevice(&deviceProp, output_verbosity);
	
	if(buffer_size_engine==0)
		buffer_size=MAX_CHUNK_SIZE;
	else 
		buffer_size=buffer_size_engine;
	
#if CUDART_VERSION >= 2000
	*nm=deviceProp.multiProcessorCount;
#endif

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	isIntegrated=deviceProp.integrated;
	if(isIntegrated) {
        	//zero-copy memory mode - use special function to get OS-pinned memory
		_CUDA(hipSetDeviceFlags(hipDeviceMapHost));
        	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using zero-copy memory.\n");
        	_CUDA(hipHostAlloc((void**)&des_h_s,buffer_size,hipHostMallocMapped));
		transferHostToDevice = transferHostToDevice_ZEROCOPY;		// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_ZEROCOPY;		// set memory transfer function
		_CUDA(hipHostGetDevicePointer(&des_d_s,des_h_s, 0));
	} else {
		//pinned memory mode - use special function to get OS-pinned memory
		_CUDA(hipHostAlloc( (void**)&des_h_s, buffer_size, hipHostMallocDefault));
		if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
		transferHostToDevice = transferHostToDevice_PINNED;	// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_PINNED;	// set memory transfer function
		_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
	}
#else
        //pinned memory mode - use special function to get OS-pinned memory
        _CUDA(hipHostMalloc((void**)&h_s, buffer_size));
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
	transferHostToDevice = transferHostToDevice_PINNED;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PINNED;			// set memory transfer function
	_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
#endif
#else
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pageable memory.\n");
	transferHostToDevice = transferHostToDevice_PAGEABLE;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PAGEABLE;			// set memory transfer function
	_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
#endif

	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"The current buffer size is %d.\n\n", buffer_size);

	_CUDA(hipEventCreate(&des_start));
	_CUDA(hipEventCreate(&des_stop));
	_CUDA(hipEventRecord(des_start,0));

}
//
// CBC parallel decrypt
//

__global__ void DESdecKernel_cbc(uint32_t in[],uint32_t out[],uint32_t iv[]) {
}

extern "C" void DES_cuda_transfer_iv(const unsigned char *iv) {
}

extern "C" void DES_cuda_decrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}

#ifndef CBC_ENC_CPU
//
// CBC  encrypt
//

__global__ void DESencKernel_cbc(uint32_t state[],uint32_t iv[],size_t length) {
}

#endif

extern "C" void DES_cuda_encrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}
#else
#error "ERROR: DEVICE EMULATION is NOT supported."
#endif
