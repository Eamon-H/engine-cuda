#include "hip/hip_runtime.h"
#ifndef __DEVICE_EMULATION__

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <openssl/des.h>
#include "cuda_common.h"
#include "common.h"
#include "cuPrintf.cu"

//#define MAX_THREAD		256
//#define STATE_THREAD_DES	2	

//#define DES_MAXNR		8
//#define DES_BLOCK_SIZE		8
//#define DES_KEY_SIZE		8

__constant__ uint32_t des_d_sp[8][64]={
{
/* nibble 0 */
0x02080800L, 0x00080000L, 0x02000002L, 0x02080802L,
0x02000000L, 0x00080802L, 0x00080002L, 0x02000002L,
0x00080802L, 0x02080800L, 0x02080000L, 0x00000802L,
0x02000802L, 0x02000000L, 0x00000000L, 0x00080002L,
0x00080000L, 0x00000002L, 0x02000800L, 0x00080800L,
0x02080802L, 0x02080000L, 0x00000802L, 0x02000800L,
0x00000002L, 0x00000800L, 0x00080800L, 0x02080002L,
0x00000800L, 0x02000802L, 0x02080002L, 0x00000000L,
0x00000000L, 0x02080802L, 0x02000800L, 0x00080002L,
0x02080800L, 0x00080000L, 0x00000802L, 0x02000800L,
0x02080002L, 0x00000800L, 0x00080800L, 0x02000002L,
0x00080802L, 0x00000002L, 0x02000002L, 0x02080000L,
0x02080802L, 0x00080800L, 0x02080000L, 0x02000802L,
0x02000000L, 0x00000802L, 0x00080002L, 0x00000000L,
0x00080000L, 0x02000000L, 0x02000802L, 0x02080800L,
0x00000002L, 0x02080002L, 0x00000800L, 0x00080802L,
},{
/* nibble 1 */
0x40108010L, 0x00000000L, 0x00108000L, 0x40100000L,
0x40000010L, 0x00008010L, 0x40008000L, 0x00108000L,
0x00008000L, 0x40100010L, 0x00000010L, 0x40008000L,
0x00100010L, 0x40108000L, 0x40100000L, 0x00000010L,
0x00100000L, 0x40008010L, 0x40100010L, 0x00008000L,
0x00108010L, 0x40000000L, 0x00000000L, 0x00100010L,
0x40008010L, 0x00108010L, 0x40108000L, 0x40000010L,
0x40000000L, 0x00100000L, 0x00008010L, 0x40108010L,
0x00100010L, 0x40108000L, 0x40008000L, 0x00108010L,
0x40108010L, 0x00100010L, 0x40000010L, 0x00000000L,
0x40000000L, 0x00008010L, 0x00100000L, 0x40100010L,
0x00008000L, 0x40000000L, 0x00108010L, 0x40008010L,
0x40108000L, 0x00008000L, 0x00000000L, 0x40000010L,
0x00000010L, 0x40108010L, 0x00108000L, 0x40100000L,
0x40100010L, 0x00100000L, 0x00008010L, 0x40008000L,
0x40008010L, 0x00000010L, 0x40100000L, 0x00108000L,
},{
/* nibble 2 */
0x04000001L, 0x04040100L, 0x00000100L, 0x04000101L,
0x00040001L, 0x04000000L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00040000L, 0x04040000L, 0x00000001L,
0x04040101L, 0x00000101L, 0x00000001L, 0x04040001L,
0x00000000L, 0x00040001L, 0x04040100L, 0x00000100L,
0x00000101L, 0x04040101L, 0x00040000L, 0x04000001L,
0x04040001L, 0x04000100L, 0x00040101L, 0x04040000L,
0x00040100L, 0x00000000L, 0x04000000L, 0x00040101L,
0x04040100L, 0x00000100L, 0x00000001L, 0x00040000L,
0x00000101L, 0x00040001L, 0x04040000L, 0x04000101L,
0x00000000L, 0x04040100L, 0x00040100L, 0x04040001L,
0x00040001L, 0x04000000L, 0x04040101L, 0x00000001L,
0x00040101L, 0x04000001L, 0x04000000L, 0x04040101L,
0x00040000L, 0x04000100L, 0x04000101L, 0x00040100L,
0x04000100L, 0x00000000L, 0x04040001L, 0x00000101L,
0x04000001L, 0x00040101L, 0x00000100L, 0x04040000L,
},{
/* nibble 3 */
0x00401008L, 0x10001000L, 0x00000008L, 0x10401008L,
0x00000000L, 0x10400000L, 0x10001008L, 0x00400008L,
0x10401000L, 0x10000008L, 0x10000000L, 0x00001008L,
0x10000008L, 0x00401008L, 0x00400000L, 0x10000000L,
0x10400008L, 0x00401000L, 0x00001000L, 0x00000008L,
0x00401000L, 0x10001008L, 0x10400000L, 0x00001000L,
0x00001008L, 0x00000000L, 0x00400008L, 0x10401000L,
0x10001000L, 0x10400008L, 0x10401008L, 0x00400000L,
0x10400008L, 0x00001008L, 0x00400000L, 0x10000008L,
0x00401000L, 0x10001000L, 0x00000008L, 0x10400000L,
0x10001008L, 0x00000000L, 0x00001000L, 0x00400008L,
0x00000000L, 0x10400008L, 0x10401000L, 0x00001000L,
0x10000000L, 0x10401008L, 0x00401008L, 0x00400000L,
0x10401008L, 0x00000008L, 0x10001000L, 0x00401008L,
0x00400008L, 0x00401000L, 0x10400000L, 0x10001008L,
0x00001008L, 0x10000000L, 0x10000008L, 0x10401000L,
},{
/* nibble 4 */
0x08000000L, 0x00010000L, 0x00000400L, 0x08010420L,
0x08010020L, 0x08000400L, 0x00010420L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x00010400L,
0x08000420L, 0x08010020L, 0x08010400L, 0x00000000L,
0x00010400L, 0x08000000L, 0x00010020L, 0x00000420L,
0x08000400L, 0x00010420L, 0x00000000L, 0x08000020L,
0x00000020L, 0x08000420L, 0x08010420L, 0x00010020L,
0x08010000L, 0x00000400L, 0x00000420L, 0x08010400L,
0x08010400L, 0x08000420L, 0x00010020L, 0x08010000L,
0x00010000L, 0x00000020L, 0x08000020L, 0x08000400L,
0x08000000L, 0x00010400L, 0x08010420L, 0x00000000L,
0x00010420L, 0x08000000L, 0x00000400L, 0x00010020L,
0x08000420L, 0x00000400L, 0x00000000L, 0x08010420L,
0x08010020L, 0x08010400L, 0x00000420L, 0x00010000L,
0x00010400L, 0x08010020L, 0x08000400L, 0x00000420L,
0x00000020L, 0x00010420L, 0x08010000L, 0x08000020L,
},{
/* nibble 5 */
0x80000040L, 0x00200040L, 0x00000000L, 0x80202000L,
0x00200040L, 0x00002000L, 0x80002040L, 0x00200000L,
0x00002040L, 0x80202040L, 0x00202000L, 0x80000000L,
0x80002000L, 0x80000040L, 0x80200000L, 0x00202040L,
0x00200000L, 0x80002040L, 0x80200040L, 0x00000000L,
0x00002000L, 0x00000040L, 0x80202000L, 0x80200040L,
0x80202040L, 0x80200000L, 0x80000000L, 0x00002040L,
0x00000040L, 0x00202000L, 0x00202040L, 0x80002000L,
0x00002040L, 0x80000000L, 0x80002000L, 0x00202040L,
0x80202000L, 0x00200040L, 0x00000000L, 0x80002000L,
0x80000000L, 0x00002000L, 0x80200040L, 0x00200000L,
0x00200040L, 0x80202040L, 0x00202000L, 0x00000040L,
0x80202040L, 0x00202000L, 0x00200000L, 0x80002040L,
0x80000040L, 0x80200000L, 0x00202040L, 0x00000000L,
0x00002000L, 0x80000040L, 0x80002040L, 0x80202000L,
0x80200000L, 0x00002040L, 0x00000040L, 0x80200040L,
},{
/* nibble 6 */
0x00004000L, 0x00000200L, 0x01000200L, 0x01000004L,
0x01004204L, 0x00004004L, 0x00004200L, 0x00000000L,
0x01000000L, 0x01000204L, 0x00000204L, 0x01004000L,
0x00000004L, 0x01004200L, 0x01004000L, 0x00000204L,
0x01000204L, 0x00004000L, 0x00004004L, 0x01004204L,
0x00000000L, 0x01000200L, 0x01000004L, 0x00004200L,
0x01004004L, 0x00004204L, 0x01004200L, 0x00000004L,
0x00004204L, 0x01004004L, 0x00000200L, 0x01000000L,
0x00004204L, 0x01004000L, 0x01004004L, 0x00000204L,
0x00004000L, 0x00000200L, 0x01000000L, 0x01004004L,
0x01000204L, 0x00004204L, 0x00004200L, 0x00000000L,
0x00000200L, 0x01000004L, 0x00000004L, 0x01000200L,
0x00000000L, 0x01000204L, 0x01000200L, 0x00004200L,
0x00000204L, 0x00004000L, 0x01004204L, 0x01000000L,
0x01004200L, 0x00000004L, 0x00004004L, 0x01004204L,
0x01000004L, 0x01004200L, 0x01004000L, 0x00004004L,
},{
/* nibble 7 */
0x20800080L, 0x20820000L, 0x00020080L, 0x00000000L,
0x20020000L, 0x00800080L, 0x20800000L, 0x20820080L,
0x00000080L, 0x20000000L, 0x00820000L, 0x00020080L,
0x00820080L, 0x20020080L, 0x20000080L, 0x20800000L,
0x00020000L, 0x00820080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x20000080L, 0x00000000L, 0x00820000L,
0x20000000L, 0x00800000L, 0x20020080L, 0x20800080L,
0x00800000L, 0x00020000L, 0x20820000L, 0x00000080L,
0x00800000L, 0x00020000L, 0x20000080L, 0x20820080L,
0x00020080L, 0x20000000L, 0x00000000L, 0x00820000L,
0x20800080L, 0x20020080L, 0x20020000L, 0x00800080L,
0x20820000L, 0x00000080L, 0x00800080L, 0x20020000L,
0x20820080L, 0x00800000L, 0x20800000L, 0x20000080L,
0x00820000L, 0x00020080L, 0x20020080L, 0x20800000L,
0x00000080L, 0x20820000L, 0x00820080L, 0x00000000L,
0x20000000L, 0x20800080L, 0x00020000L, 0x00820080L,
}};

__constant__ uint32_t des_rk[32];

__device__ __shared__ uint32_t *des_d_s;
//__device__ uint32_t *des_d_iv;
//__device__ uint32_t *des_d_out;

uint8_t  *des_h_s;
//uint8_t  *des_h_out;
//uint8_t  *des_h_iv;

float des_elapsed;
hipEvent_t des_start,des_stop;

#define IP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(right,left,tt, 4,0x0f0f0f0fL); \
	PERM_OP(left,right,tt,16,0x0000ffffL); \
	PERM_OP(right,left,tt, 2,0x33333333L); \
	PERM_OP(left,right,tt, 8,0x00ff00ffL); \
	PERM_OP(right,left,tt, 1,0x55555555L); \
	}

#define FP(left,right) \
	{ \
	register uint32_t tt; \
	PERM_OP(left,right,tt, 1,0x55555555L); \
	PERM_OP(right,left,tt, 8,0x00ff00ffL); \
	PERM_OP(left,right,tt, 2,0x33333333L); \
	PERM_OP(right,left,tt,16,0x0000ffffL); \
	PERM_OP(left,right,tt, 4,0x0f0f0f0fL); \
	}

#define	ROTATE(a,n)	(((a)>>(n))|((a)<<(32-(n))))

#define PERM_OP(a,b,t,n,m) ((t)=((((a)>>(n))^(b))&(m)),\
	(b)^=(t),\
	(a)^=((t)<<(n)))

#define D_ENCRYPT(LL,R,S) { \
	u=R^s[S  ]; \
	t=R^s[S+1]; \
	t=ROTATE(t,4); \
	LL^= \
	*(const uint32_t *)(des_SP      +((u     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x200+((u>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x400+((u>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x600+((u>>24L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x100+((t     )&0xfc))^ \
	*(const uint32_t *)(des_SP+0x300+((t>> 8L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x500+((t>>16L)&0xfc))^ \
	*(const uint32_t *)(des_SP+0x700+((t>>24L)&0xfc)); }

__global__ void DESencKernel(uint32_t *data, uint32_t *s) {
	uint32_t tx = blockIdx.x * (blockDim.x * blockDim.y) + (blockDim.y * threadIdx.x) + threadIdx.y;
	
	uint32_t right = data[2*tx];
	uint32_t left = data[2*tx+1];

	unsigned int t,u;
	unsigned char *des_SP = (unsigned char *) (&des_d_sp);

	s=des_rk;

	IP(right,left);

	left=ROTATE(left,29);
	right=ROTATE(right,29);

	D_ENCRYPT(left,right, 0); /*  1 */
	D_ENCRYPT(right,left, 2); /*  2 */
	D_ENCRYPT(left,right, 4); /*  3 */
	D_ENCRYPT(right,left, 6); /*  4 */
	D_ENCRYPT(left,right, 8); /*  5 */
	D_ENCRYPT(right,left,10); /*  6 */
	D_ENCRYPT(left,right,12); /*  7 */
	D_ENCRYPT(right,left,14); /*  8 */
	D_ENCRYPT(left,right,16); /*  9 */
	D_ENCRYPT(right,left,18); /*  10 */
	D_ENCRYPT(left,right,20); /*  11 */
	D_ENCRYPT(right,left,22); /*  12 */
	D_ENCRYPT(left,right,24); /*  13 */
	D_ENCRYPT(right,left,26); /*  14 */
	D_ENCRYPT(left,right,28); /*  15 */
	D_ENCRYPT(right,left,30); /*  16 */

	left=ROTATE(left,3);
	right=ROTATE(right,3);

	FP(right,left);
	data[2*tx]=left;
	data[2*tx+1]=right;
}

__global__ void DESdecKernel(uint32_t *data) {
}

extern "C" void DES_cuda_encrypt(const unsigned char *in, unsigned char *out, size_t nbytes) {

	hipError_t cudaerrno;
	assert(in && out && nbytes);
	int gridSize = nbytes/MAX_THREAD+1;
	dim3 dimBlock(MAX_THREAD, 1, 1);

	transferHostToDevice(&in, &des_d_s, &des_h_s, &nbytes);

	if ((nbytes%(MAX_THREAD*DES_BLOCK_SIZE))==0) {
		//gridSize = nbytes/(MAX_THREAD*DES_BLOCK_SIZE);
		//dimBlock.x = nbytes/DES_BLOCK_SIZE;
	} else {
		dimBlock.x = MAX_THREAD;
	}

	if (output_verbosity==OUTPUT_VERBOSE)
		fprintf(stdout,"Starting DES kernel with (%d, (%d, %d))...\n", gridSize, dimBlock.x, dimBlock.y);

	DESencKernel<<<gridSize,dimBlock>>>(des_d_s, des_rk);
	_CUDA_N("DES encryption kernel could not be launched!");

	transferDeviceToHost(&out, &des_d_s, &des_h_s, &nbytes);
}

extern "C" void DES_cuda_decrypt(const unsigned char *in, unsigned char *out,size_t nbytes) {
	assert(in && out && nbytes);
	hipError_t cudaerrno;
	int gridSize = 1;
	dim3 dimBlock(DES_BLOCK_SIZE, nbytes/DES_BLOCK_SIZE, 1);

	if (output_verbosity==OUTPUT_VERBOSE) {
		fprintf(stdout,"\nDES Size: %d\n",(int)nbytes);
		fprintf(stdout,"Starting DES decryption...");
	}

	transferHostToDevice(&in, &des_d_s, &des_h_s, &nbytes);

	if (output_verbosity==OUTPUT_VERBOSE)
		fprintf(stdout,"DES kernel execution...");

	if ((nbytes%(MAX_THREAD*DES_BLOCK_SIZE))==0) {
		gridSize = nbytes/(MAX_THREAD*DES_BLOCK_SIZE);
		dimBlock.y = MAX_THREAD/DES_BLOCK_SIZE;
	} else {
		dimBlock.y= 1024/DES_BLOCK_SIZE;
	}

	DESdecKernel<<<gridSize,dimBlock>>>(des_d_s);
	_CUDA_N("DES decryption kernel could not be launched!");

	transferDeviceToHost(&out, &des_d_s, &des_h_s, &nbytes);
}

extern "C" void DES_cuda_transfer_key_schedule(DES_key_schedule *ks) {
	assert(ks);
	size_t ks_size = sizeof(DES_key_schedule);
	hipMemcpyToSymbol(HIP_SYMBOL(des_rk),ks,ks_size,0,hipMemcpyHostToDevice);
}

extern "C" void DES_cuda_finish() {
	hipError_t cudaerrno;

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout, "\nDone. Finishing up DES\n");

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	if(isIntegrated) {
		_CUDA(hipHostFree(des_h_s));
		//_CUDA(hipHostFree(des_h_out));
		//_CUDA(hipHostFree(des_h_iv));
	} else {
		_CUDA(hipFree(des_d_s));
		//_CUDA(hipFree(des_d_out));
		//_CUDA(hipFree(des_d_iv));
	}
#else	
	_CUDA(hipFree(des_d_s));
	//_CUDA(hipFree(des_d_out));
	//_CUDA(hipFree(des_d_iv));
#endif
#else
	_CUDA(hipFree(des_d_s));
	//_CUDA(hipFree(des_d_out));
	//_CUDA(hipFree(des_d_iv));
#endif	

	_CUDA(hipEventRecord(des_stop,0));
	_CUDA(hipEventSynchronize(des_stop));
	_CUDA(hipEventElapsedTime(&des_elapsed,des_start,des_stop));

	if (output_verbosity>=OUTPUT_NORMAL) fprintf(stdout,"\nTotal time: %f milliseconds\n",des_elapsed);	
}

extern "C" void DES_cuda_init(int *nm, int buffer_size_engine, int output_kind) {
	assert(nm);
	hipError_t cudaerrno;
   	int buffer_size;
	hipDeviceProp_t deviceProp;
    	
	output_verbosity=output_kind;

	checkCUDADevice(&deviceProp, output_verbosity);
	
	if(buffer_size_engine==0)
		buffer_size=MAX_CHUNK_SIZE;
	else 
		buffer_size=buffer_size_engine;
	
#if CUDART_VERSION >= 2000
	*nm=deviceProp.multiProcessorCount;
#endif

#ifndef PAGEABLE 
#if CUDART_VERSION >= 2020
	isIntegrated=deviceProp.integrated;
	if(isIntegrated) {
        	//zero-copy memory mode - use special function to get OS-pinned memory
		_CUDA(hipSetDeviceFlags(hipDeviceMapHost));
        	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using zero-copy memory.\n");
        	_CUDA(hipHostAlloc((void**)&des_h_s,buffer_size,hipHostMallocMapped));
		//_CUDA(hipHostAlloc((void**)&des_h_out,buffer_size,hipHostMallocMapped));
		//_CUDA(hipHostAlloc((void**)&des_h_iv,buffer_size,hipHostMallocMapped));
		transferHostToDevice = transferHostToDevice_ZEROCOPY;		// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_ZEROCOPY;		// set memory transfer function
		_CUDA(hipHostGetDevicePointer(&des_d_s,des_h_s, 0));
		//_CUDA(hipHostGetDevicePointer(&des_d_out,des_h_out, 0));
		//_CUDA(hipHostGetDevicePointer(&des_d_iv,des_h_iv, 0));
	} else {
		//pinned memory mode - use special function to get OS-pinned memory
		_CUDA(hipHostAlloc( (void**)&des_h_s, buffer_size, hipHostMallocDefault));
		//_CUDA(hipHostAlloc( (void**)&des_h_out, buffer_size, hipHostMallocDefault));
		//_CUDA(hipHostAlloc( (void**)&des_h_iv, buffer_size, hipHostMallocDefault));
		if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
		transferHostToDevice = transferHostToDevice_PINNED;	// set memory transfer function
		transferDeviceToHost = transferDeviceToHost_PINNED;	// set memory transfer function
		_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
		//_CUDA(hipMalloc((void **)&des_d_out,buffer_size));
		//_CUDA(hipMalloc((void **)&des_d_iv,DES_BLOCK_SIZE));
	}
#else
        //pinned memory mode - use special function to get OS-pinned memory
        _CUDA(hipHostMalloc((void**)&h_s, buffer_size));
        //_CUDA(hipHostMalloc((void**)&h_out, buffer_size));
        //_CUDA(hipHostMalloc((void**)&h_iv, buffer_size));
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pinned memory: hipHostMallocDefault.\n");
	transferHostToDevice = transferHostToDevice_PINNED;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PINNED;			// set memory transfer function
	_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
	//_CUDA(hipMalloc((void **)&des_d_out,buffer_size));
        //_CUDA(hipMalloc((void **)&des_d_iv,DES_BLOCK_SIZE));
#endif
#else
        if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"Using pageable memory.\n");
	transferHostToDevice = transferHostToDevice_PAGEABLE;			// set memory transfer function
	transferDeviceToHost = transferDeviceToHost_PAGEABLE;			// set memory transfer function
	_CUDA(hipMalloc((void **)&des_d_s,buffer_size));
	//_CUDA(hipMalloc((void **)&des_d_out,buffer_size));
        //_CUDA(hipMalloc((void **)&des_d_iv,DES_BLOCK_SIZE));
#endif

	if (output_verbosity!=OUTPUT_QUIET) fprintf(stdout,"The current buffer size is %d.\n\n", buffer_size);
	_CUDA(hipMalloc((void **)&des_rk, (int) sizeof(DES_key_schedule)));

	_CUDA(hipEventCreate(&des_start));
	_CUDA(hipEventCreate(&des_stop));
	_CUDA(hipEventRecord(des_start,0));

}
//
// CBC parallel decrypt
//

__global__ void DESdecKernel_cbc(uint32_t in[],uint32_t out[],uint32_t iv[]) {
}

extern "C" void DES_cuda_transfer_iv(const unsigned char *iv) {
}

extern "C" void DES_cuda_decrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}

#ifndef CBC_ENC_CPU
//
// CBC  encrypt
//

__global__ void DESencKernel_cbc(uint32_t state[],uint32_t iv[],size_t length) {
}

#endif

extern "C" void DES_cuda_encrypt_cbc(const unsigned char *in, unsigned char *out, size_t nbytes) {
}
#else
#error "ERROR: DEVICE EMULATION is NOT supported."
#endif
